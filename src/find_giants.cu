/***************************************************************************
 *
 *   Copyright (C) 2012 by Ben Barsdell and Andrew Jameson
 *   Licensed under the Academic Free License version 2.1
 *
 ***************************************************************************/

#include <hd/find_giants.hpp>

// TESTING only
#include <utils/stopwatch.hpp>
#include <fmt/format.h>
//#define PRINT_BENCHMARKS

#include <thrust/device_vector.h>
#include <thrust/count.h>
#include <thrust/copy.h>
#include <thrust/adjacent_difference.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <thrust/binary_search.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>

template <typename T>
struct greater_than_val : public thrust::unary_function<T, bool> {
    T val;
    greater_than_val(T val_) : val(val_) {}
    inline __host__ __device__ bool operator()(T x) const { return x > val; }
};

template <typename T>
struct maximum_first : public thrust::binary_function<T, T, T> {
    inline __host__ __device__ T operator()(T a, T b) const {
        return thrust::get<0>(a) >= thrust::get<0>(b) ? a : b;
    }
};

template <typename T>
struct nearby : public thrust::binary_function<T, T, bool> {
    T max_dist;
    nearby(T max_dist_) : max_dist(max_dist_) {}
    inline __host__ __device__ bool operator()(T a, T b) const {
        return b <= a + max_dist;
    }
};
template <typename T>
struct not_nearby : public thrust::binary_function<T, T, bool> {
    T max_dist;
    not_nearby(T max_dist_) : max_dist(max_dist_) {}
    inline __host__ __device__ bool operator()(T b, T a) const {
        return b > a + max_dist;
    }
};

template <typename T>
struct plus_one : public thrust::unary_function<T, T> {
    inline __host__ __device__ T operator()(T x) const { return x + 1; }
};

class GiantFinder_impl {
    thrust::device_vector<hd_float> d_giant_data;
    thrust::device_vector<hd_size> d_giant_data_inds;
    thrust::device_vector<int> d_giant_data_segments;
    thrust::device_vector<hd_size> d_giant_data_seg_ids;

public:
    hd_error exec(const hd_float* d_data, hd_size count, hd_float thresh,
                  hd_size merge_dist,
                  thrust::device_vector<hd_float>& d_giant_peaks,
                  thrust::device_vector<hd_size>& d_giant_inds,
                  thrust::device_vector<hd_size>& d_giant_begins,
                  thrust::device_vector<hd_size>& d_giant_ends,
                  cached_allocator& policy) {
        // This algorithm works by extracting all samples in the time series
        //   above thresh (the giant_data), segmenting those samples into
        //   isolated giants (based on merge_dist), and then computing the
        //   details of each giant into the d_giant_* arrays using
        //   reduce_by_key and some scatter operations.

        typedef thrust::device_ptr<const hd_float> const_float_ptr;
        typedef thrust::device_ptr<hd_float> float_ptr;
        typedef thrust::device_ptr<hd_size> size_ptr;

        const_float_ptr d_data_begin(d_data);
        const_float_ptr d_data_end(d_data + count);

#ifdef PRINT_BENCHMARKS
        Stopwatch timer;
        timer.start();
#endif

        // Note: Thrust functions are called by passing policy through
        //       cuda::par as the first parameter to cause allocations to be
        //       handled by custom cached allocator `policy`.
        //       This turns out to be critical to performance!

        // Quickly count how much giant data there is so we know the space
        // needed
        hd_size giant_data_count
            = thrust::count_if(thrust::cuda::par(policy), d_data_begin,
                               d_data_end, greater_than_val<hd_float>(thresh));
        // We can bail early if there are no giants at all
        if (0 == giant_data_count) {
            return HD_NO_ERROR;
        }

#ifdef PRINT_BENCHMARKS
        hipDeviceSynchronize();
        timer.stop();
        fmt::print("{:<25}: {} s\n", "count_if time", timer.getTime());
        timer.reset();
        timer.start();
#endif

        d_giant_data.resize(giant_data_count);
        d_giant_data_inds.resize(giant_data_count);

#ifdef PRINT_BENCHMARKS
        hipDeviceSynchronize();
        timer.stop();
        fmt::print("{:<25}: {} s\n", "giant_data resize time", timer.getTime());
        timer.reset();
        timer.start();
#endif

        // Copy all of the giant data and their locations into one place
        hd_size giant_data_count2
            = thrust::copy_if(
                  thrust::cuda::par(policy),
                  thrust::make_zip_iterator(thrust::make_tuple(
                      d_data_begin, thrust::make_counting_iterator(0u))),
                  thrust::make_zip_iterator(thrust::make_tuple(
                      d_data_begin, thrust::make_counting_iterator(0u)))
                      + count,
                  (d_data_begin),  // the stencil
                  thrust::make_zip_iterator(thrust::make_tuple(
                      d_giant_data.begin(), d_giant_data_inds.begin())),
                  greater_than_val<hd_float>(thresh))
              - thrust::make_zip_iterator(thrust::make_tuple(
                  d_giant_data.begin(), d_giant_data_inds.begin()));

#ifdef PRINT_BENCHMARKS
        hipDeviceSynchronize();
        timer.stop();
        fmt::print("{:<25}: {} s\n", "giant_data copy_if time",
                   timer.getTime());
        timer.reset();
        timer.start();
#endif

        // Create an array of head flags indicating candidate segments
        d_giant_data_segments.resize(giant_data_count);
        thrust::adjacent_difference(
            thrust::cuda::par(policy), d_giant_data_inds.begin(),
            d_giant_data_inds.end(), d_giant_data_segments.begin(),
            not_nearby<hd_size>(merge_dist));

        // The first element is implicitly a segment head
        if (giant_data_count > 0) {
            d_giant_data_segments.front() = 0;
        }

        d_giant_data_seg_ids.resize(d_giant_data_segments.size());
        thrust::inclusive_scan(
            thrust::cuda::par(policy), d_giant_data_segments.begin(),
            d_giant_data_segments.end(), d_giant_data_seg_ids.begin());

        // We extract the number of giants from the end of the exclusive scan
        hd_size giant_count = d_giant_data_seg_ids.back() + 1;

#ifdef PRINT_BENCHMARKS
        hipDeviceSynchronize();
        timer.stop();
        fmt::print("{:<25}: {} s\n", "giant segments time", timer.getTime());
        timer.reset();
        timer.start();
#endif

        hd_size new_giants_offset = d_giant_peaks.size();
        // Allocate space for the new giants
        d_giant_peaks.resize(d_giant_peaks.size() + giant_count);
        d_giant_inds.resize(d_giant_inds.size() + giant_count);
        d_giant_begins.resize(d_giant_begins.size() + giant_count);
        d_giant_ends.resize(d_giant_ends.size() + giant_count);
        float_ptr new_giant_peaks_begin(&d_giant_peaks[new_giants_offset]);
        size_ptr new_giant_inds_begin(&d_giant_inds[new_giants_offset]);
        size_ptr new_giant_begins_begin(&d_giant_begins[new_giants_offset]);
        size_ptr new_giant_ends_begin(&d_giant_ends[new_giants_offset]);

#ifdef PRINT_BENCHMARKS
        hipDeviceSynchronize();
        timer.stop();
        fmt::print("{:<25}: {} s\n", "giants resize time", timer.getTime());
        timer.reset();
        timer.start();
#endif

        // Now we find the value (snr) and location (time) of each giant's
        // maximum
        hd_size giant_count2
            = reduce_by_key(
                  thrust::cuda::par(policy),
                  d_giant_data_inds.begin(),  // the keys
                  d_giant_data_inds.end(),
                  thrust::make_zip_iterator(thrust::make_tuple(
                      d_giant_data.begin(), d_giant_data_inds.begin())),
                  thrust::make_discard_iterator(),  // the keys output
                  thrust::make_zip_iterator(thrust::make_tuple(
                      new_giant_peaks_begin, new_giant_inds_begin)),
                  nearby<hd_size>(merge_dist),
                  maximum_first<thrust::tuple<hd_float, hd_size>>())
                  .second
              - thrust::make_zip_iterator(thrust::make_tuple(
                  new_giant_peaks_begin, new_giant_inds_begin));

#ifdef PRINT_BENCHMARKS
        hipDeviceSynchronize();
        timer.stop();
        fmt::print("{:<25}: {} s\n", "reduce_by_key time", timer.getTime());
        timer.reset();
        timer.start();
#endif

        // Now we make the first segment explicit
        if (giant_count > 0) {
            d_giant_data_segments[0] = 1;
        }

        // Create arrays of the beginning and end indices of each giant
        thrust::scatter_if(d_giant_data_inds.begin(), d_giant_data_inds.end(),
                           d_giant_data_seg_ids.begin(),
                           d_giant_data_segments.begin(),
                           new_giant_begins_begin);
        thrust::scatter_if(make_transform_iterator(d_giant_data_inds.begin(),
                                                   plus_one<hd_size>()),
                           make_transform_iterator(d_giant_data_inds.end() - 1,
                                                   plus_one<hd_size>()),
                           d_giant_data_seg_ids.begin(),
                           d_giant_data_segments.begin() + 1,
                           new_giant_ends_begin);

        if (giant_count > 0) {
            d_giant_ends.back() = d_giant_data_inds.back() + 1;
        }

#ifdef PRINT_BENCHMARKS
        hipDeviceSynchronize();
        timer.stop();
        fmt::print("{:<25}: {} s\n", "begin/end copy_if time", timer.getTime());
        timer.reset();
        fmt::print("--------------------");
#endif

        return HD_NO_ERROR;
    }
};

// Public interface (wrapper for implementation)
GiantFinder::GiantFinder() : m_impl(new GiantFinder_impl) {}
hd_error GiantFinder::exec(const hd_float* d_data, hd_size count,
                           hd_float thresh, hd_size merge_dist,
                           thrust::device_vector<hd_float>& d_giant_peaks,
                           thrust::device_vector<hd_size>& d_giant_inds,
                           thrust::device_vector<hd_size>& d_giant_begins,
                           thrust::device_vector<hd_size>& d_giant_ends) {
    return m_impl->exec(d_data, count, thresh, merge_dist, d_giant_peaks,
                        d_giant_inds, d_giant_begins, d_giant_ends, allocator);
}
